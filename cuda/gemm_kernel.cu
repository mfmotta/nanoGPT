#include "hip/hip_runtime.h"
/*
  This kernel is based on Cutlass-1.3 sgemm: 
  https://github.com/NVIDIA/cutlass/blob/master/examples/00_basic_gemm/basic_gemm.cu

  The CUTLASS Gemm template is instantiated in the function CutlassSgemmNN. This kernel:
  
  > computes the general matrix product (GEMM) using single-precision floating-point arithmetic
  > assumes all matrices have column-major layout.

  Threadblock tile size is chosen to be 128x128x8 

  To view the full gemm device API interface, see `cutlass/gemm/device/gemm.h` 
  (https://github.com/NVIDIA/cutlass/blob/master/include/cutlass/gemm/device/gemm.h)
*/

#include <iostream>
#include <sstream>
#include <vector>
#include "helper.h"
#include "cutlass/gemm/device/gemm.h"

// This function defines a CUTLASS GEMM kernel instantiation, constructs its parameters object,
// and launches it on the CUDA device.

/// Define a CUTLASS GEMM template and launch a GEMM kernel.
hipError_t CutlassSgemmNN(
    int M,
    int N,
    int K,
    float alpha,
    float const *A,
    int lda,
    float const *B,
    int ldb,
    float beta,
    float *C,
    int ldc) {

    //MM at compile time: maps data types and high-level structural parameters onto specific CUTLASS components
    using ColumnMajor = cutlass::layout::ColumnMajor;

    using CutlassGemm = cutlass::gemm::device::Gemm<float,        // Data-type of A matrix
                                                    ColumnMajor,  // Layout of A matrix
                                                    float,        // Data-type of B matrix
                                                    ColumnMajor,  // Layout of B matrix
                                                    float,        // Data-type of C matrix
                                                    ColumnMajor>; // Layout of C matrix

    // Define a CUTLASS GEMM type
    CutlassGemm gemm_operator;

    // Construct the CUTLASS GEMM arguments object.
    //
    // gemm argument objects are constructible in host code and passed to kernels by value. 
    // These may include pointers, strides, scalars, and other arguments needed by Gemm and its components.
    //
    // Benefits of this pattern: (1.) structured, composable strategy for passing host-constructible
    // arguments to kernels and (2.) minimized initialization overhead on kernel entry.

    // MM At runtime, map logical arguments to GEMM problems to kernel parameters.
    CutlassGemm::Arguments args({M , N, K},  // Gemm Problem dimensions
                                {A, lda},    // Tensor-ref for source matrix A TODO:?with leading dimension lda=column (col major)
                                {B, ldb},    // Tensor-ref for source matrix B
                                {C, ldc},    // Tensor-ref for source matrix C
                                {C, ldc},    // Tensor-ref for destination matrix D (may be different memory than source C matrix)
                                {alpha, beta}); // Scalars used in the Epilogue

    // Launch the CUTLASS GEMM kernel on the device at runtime.
    cutlass::Status status = gemm_operator(args);

    // Return a hipError_t if the CUTLASS GEMM operator returned an error code.
    if (status != cutlass::Status::kSuccess) {
        return hipErrorUnknown;
    }

    // Return success, if no errors were encountered.
    return hipSuccess;
    }


// The source code after this point in the file is generic CUDA using the CUDA Runtime API
// and simple CUDA kernels to initialize matrices and compute the general matrix product.

/// Kernel to initialize a matrix with small integers.
__global__ void InitializeMatrix_kernel(
  float *matrix,
  int rows,
  int columns,
  int seed = 0) {

  int i = threadIdx.x + blockIdx.x * blockDim.x;
  int j = threadIdx.y + blockIdx.y * blockDim.y;

  if (i < rows && j < columns) {
    int offset = i + j * rows;

    // Generate arbitrary elements.
    int const k = 16807;
    int const m = 16;
    float value = float(((offset + seed) * k % m) - m / 2);

    matrix[offset] = value;
  }
}

// Function that uses kernel to initialize a matrix to arbitrary small integers.
hipError_t InitializeMatrix(float *matrix, int rows, int columns, int seed = 0) {

  dim3 block(16, 16);
  dim3 grid(
    (rows + block.x - 1) / block.x,
    (columns + block.y - 1) / block.y
  );

  InitializeMatrix_kernel<<< grid, block >>>(matrix, rows, columns, seed);

  return hipGetLastError();
}

// Will be used to create matrices A, B, C_cutlass, and C_reference (last two with same seed)
// Allocates device memory for a matrix then fill with arbitrary small integers.
hipError_t AllocateMatrix(float **matrix, int rows, int columns, int seed = 0) {
  hipError_t result;

  size_t sizeof_matrix = sizeof(float) * rows * columns;

  // Allocate device memory.
  result = hipMalloc(reinterpret_cast<void **>(matrix), sizeof_matrix);

  if (result != hipSuccess) {
    std::cerr << "Failed to allocate matrix: "
      << hipGetErrorString(result) << std::endl;
    return result;
  }

  // Clear the allocation.
  result = hipMemset(*matrix, 0, sizeof_matrix);

  if (result != hipSuccess) {
    std::cerr << "Failed to clear matrix device memory: "
      << hipGetErrorString(result) << std::endl;
    return result;
  }

  // Initialize matrix elements to arbitrary small integers.
  result = InitializeMatrix(*matrix, rows, columns, seed);

  if (result != hipSuccess) {
    std::cerr << "Failed to initialize matrix: "
      << hipGetErrorString(result) << std::endl;
    return result;
  }

  return result;
}



/// Reference GEMM computation.
__global__ void ReferenceGemm_kernel(
  int M,
  int N,
  int K,
  float alpha,
  float const *A,
  int lda,
  float const *B,
  int ldb,
  float beta,
  float *C,
  int ldc) {

  int i = threadIdx.x + blockIdx.x * blockDim.x;
  int j = threadIdx.y + blockIdx.y * blockDim.y;

  if (i < M && j < N) {
    float accumulator = 0;

    for (int k = 0; k < K; ++k) {
      accumulator += A[i + k * lda] * B[k + j * ldb];
    }

    C[i + j * ldc] = alpha * accumulator + beta * C[i + j * ldc];
  }
}

/// Reference GEMM computation.
hipError_t ReferenceGemm(
  int M,
  int N,
  int K,
  float alpha,
  float const *A,
  int lda,
  float const *B,
  int ldb,
  float beta,
  float *C,
  int ldc) {

  dim3 block(16, 16);
  dim3 grid(
    (M + block.x - 1) / block.x,
    (N + block.y - 1) / block.y
  );

  ReferenceGemm_kernel<<< grid, block >>>(M, N, K, alpha, A, lda, B, ldb, beta, C, ldc);

  return hipGetLastError();
}


/// Allocate several matrices in GPU device memory and call a single-precision
/// CUTLASS GEMM kernel.
hipError_t TestCutlassGemm(int M, int N, int K, float alpha, float beta) {
  hipError_t result;

  //
  // Define several matrices to be used as operands to GEMM kernels.
  //

  // Compute leading dimensions for each matrix.
  int lda = M;
  int ldb = K;
  int ldc = M;

  // Compute size in bytes of the C matrix.
  size_t sizeof_C = sizeof(float) * ldc * N;

  // Define pointers to matrices in GPU device memory.
  float *A;
  float *B;
  float *C_cutlass;
  float *C_reference;

  //
  // Allocate matrices in GPU device memory with arbitrary seeds. 
  //and free memory if allocation was not successful
  //

  result = AllocateMatrix(&A, M, K, 0);

  if (result !=  hipSuccess) {
    return result;
  }

  result = AllocateMatrix(&B, K, N, 17);

  if (result !=  hipSuccess) {
    hipFree(A);
    return result;
  }

  result = AllocateMatrix(&C_cutlass, M, N, 101);

  if (result != hipSuccess) {
    hipFree(A);
    hipFree(B);
    return result;
  }

  result = AllocateMatrix(&C_reference, M, N, 101);

  if (result != hipSuccess) {
    hipFree(A);
    hipFree(B);
    hipFree(C_cutlass);
    return result;
  }

  result = hipMemcpy(C_reference, C_cutlass, sizeof_C, hipMemcpyDeviceToDevice);

  if (result != hipSuccess) {
    std::cerr << "Failed to copy C_cutlass matrix to C_reference: "
      << hipGetErrorString(result) << std::endl;

    hipFree(C_reference);
    hipFree(C_cutlass);
    hipFree(B);
    hipFree(A);

    return result;
  }






    
TODO: eventually replace allocation with reading some matrix defined elsewhere, maybe python ?